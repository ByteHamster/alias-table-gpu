#include "Timer.cuh"

Timer::Timer() {
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
}

void Timer::start() {
    hipDeviceSynchronize();
    hipEventRecord(startEvent);
}

void Timer::stop() {
    hipEventRecord(stopEvent);
}

float Timer::elapsedMillis() {
    hipEventSynchronize(stopEvent);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startEvent, stopEvent);
    return milliseconds;
}

TableSplitTimer::TimingResult
        TableSplitTimer::TimingResult::operator + (TableSplitTimer::TimingResult other) const {
    return {sum + other.sum,
            partition + other.partition,
            prefixsum + other.prefixsum,
            split + other.split,
            pack + other.pack};
}

TableSplitTimer::TimingResult
        TableSplitTimer::TimingResult::operator / (int number) const {
    return {sum / number,
            partition / number,
            prefixsum / number,
            split / number,
            pack / number};
}

std::string TableSplitTimer::TimingResult::print(std::string append) const {
    return "Sum;" + std::to_string(sum) + ";" + append + "\n" +
           "Partition;" + std::to_string(partition) + ";" + append + "\n" +
           "Prefix sum;" + std::to_string(prefixsum) + ";" + append + "\n" +
           "Split;" + std::to_string(split) + ";" + append + "\n" +
           "Pack;" + std::to_string(pack) + ";" + append + "\n";
}

TableSplitTimer::TableSplitTimer () {
    for (hipEvent_t &event : events) {
        hipEventCreate(&event);
    }
}

TableSplitTimer::~TableSplitTimer () {
    for (hipEvent_t &event : events) {
        hipEventDestroy(event);
    }
}

void TableSplitTimer::notify(int event) {
    hipEventRecord(events[event]);
}

TableSplitTimer::TimingResult TableSplitTimer::get() {
    hipEventSynchronize(events[EVENT_NUM - 1]);
    float milliseconds[EVENT_NUM];
    for (int i = 1; i < EVENT_NUM; i++) {
        hipEventElapsedTime(&milliseconds[i], events[i - 1], events[i]);
    }
    return {milliseconds[EVENT_SUM_FINISHED],
            milliseconds[EVENT_PARTITION_FINISHED],
            milliseconds[EVENT_PREFIXSUM_FINISHED],
            milliseconds[EVENT_SPLIT_FINISHED],
            milliseconds[EVENT_PACK_FINISHED]};
}
